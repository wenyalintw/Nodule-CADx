#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Faster R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Shaoqing Ren
// ------------------------------------------------------------------
#ifdef __cplusplus
extern "C" {
#endif

#include <math.h>
#include <stdio.h>
#include <float.h>
#include "nms_kernel.h"

__device__ inline float devIoU(float const * const a, float const * const b) {
  /*
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + 1, 0.f), height = fmaxf(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
  */

  float ix0, iy0, iz0, id;
  float ix1, iy1, iz1;
  float xx0, yy0, zz0, dd;
  float xx1, yy1, zz1;
  float w, h, d;
  float inter;

  iz0 = a[1];
  iy0 = a[2];
  ix0 = a[3];
  id = a[4];
  iz0 -= id / 2; iz1 = iz0 + id;
  iy0 -= id / 2; iy1 = iy0 + id;
  ix0 -= id / 2; ix1 = ix0 + id;

  zz0 = b[1];
  yy0 = b[2];
  xx0 = b[3];
  dd = b[4];
  zz0 -= dd / 2; zz1 = zz0 + dd;
  yy0 -= dd / 2; yy1 = yy0 + dd;
  xx0 -= dd / 2; xx1 = xx0 + dd;

  zz0 = fmaxf(iz0, zz0);
  yy0 = fmaxf(iy0, yy0);
  xx0 = fmaxf(ix0, xx0);

  zz1 = fminf(iz1, zz1);
  yy1 = fminf(iy1, yy1);
  xx1 = fminf(ix1, xx1);

  w = fmaxf(0.0, xx1 - xx0);
  h = fmaxf(0.0, yy1 - yy0);
  d = fmaxf(0.0, zz1 - zz0);
  inter = w * h * d;

  float Sa = a[4] * a[4] * a[4];
  float Sb = b[4] * b[4] * b[4];

  // printf("iz0 %f, iy0 %f, ix0 %f, d %f\n", iz0, iy0, ix0, dd);
  // printf("iz1 %f, iy1 %f, ix1 %f\n", iz1, iy1, ix1);

  // printf("zz0 %f, yy0 %f, xx0 %f, id %f\n", zz0, yy0, xx0, id);
  // printf("zz1 %f, yy1 %f, xx1 %f\n", zz1, yy1, xx1);
  // printf("w %f, h %f, d %f\n", w, h, d);
  // printf("iarea: %f, ibarea: %f, inter: %f\n", Sa, Sb, inter);
  return inter / (Sa + Sb - inter);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


void _nms(int boxes_num, float * boxes_dev,
          unsigned long long * mask_dev, float nms_overlap_thresh) {

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);
}

#ifdef __cplusplus
}
#endif
