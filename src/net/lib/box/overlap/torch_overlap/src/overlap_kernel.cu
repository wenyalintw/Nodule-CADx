#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Written by Hao Tang
// ------------------------------------------------------------------
#ifdef __cplusplus
extern "C" {
#endif

#include <math.h>
#include <stdio.h>
#include <float.h>
#include "overlap_kernel.h"

__device__ inline float devIoU(float const * const a, float const * const b) {
  /*
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + 1, 0.f), height = fmaxf(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
  */

  float ix0, iy0, iz0, id;
  float ix1, iy1, iz1;
  float xx0, yy0, zz0, dd;
  float xx1, yy1, zz1;
  float w, h, d;
  float inter;

  iz0 = a[0];
  iy0 = a[1];
  ix0 = a[2];
  id = a[3];
  iz0 -= id / 2; iz1 = iz0 + id;
  iy0 -= id / 2; iy1 = iy0 + id;
  ix0 -= id / 2; ix1 = ix0 + id;

  zz0 = b[0];
  yy0 = b[1];
  xx0 = b[2];
  dd = b[3];
  zz0 -= dd / 2; zz1 = zz0 + dd;
  yy0 -= dd / 2; yy1 = yy0 + dd;
  xx0 -= dd / 2; xx1 = xx0 + dd;

  zz0 = fmaxf(iz0, zz0);
  yy0 = fmaxf(iy0, yy0);
  xx0 = fmaxf(ix0, xx0);

  zz1 = fminf(iz1, zz1);
  yy1 = fminf(iy1, yy1);
  xx1 = fminf(ix1, xx1);

  w = fmaxf(0.0, xx1 - xx0);
  h = fmaxf(0.0, yy1 - yy0);
  d = fmaxf(0.0, zz1 - zz0);
  inter = w * h * d;

  float Sa = a[3] * a[3] * a[3];
  float Sb = b[3] * b[3] * b[3];

  // printf("iz0 %f, iy0 %f, ix0 %f, d %f\n", iz0, iy0, ix0, dd);
  // printf("iz1 %f, iy1 %f, ix1 %f\n", iz1, iy1, ix1);

  // printf("zz0 %f, yy0 %f, xx0 %f, id %f\n", zz0, yy0, xx0, id);
  // printf("zz1 %f, yy1 %f, xx1 %f\n", zz1, yy1, xx1);
  // printf("w %f, h %f, d %f\n", w, h, d);
  // printf("iarea: %f, ibarea: %f, inter: %f\n", Sa, Sb, inter);
  return inter / (Sa + Sb - inter);
}

__global__ void nms_kernel(const int n_boxes1, const float* boxes1, const int n_boxes2, const float* boxes2, float* overlap) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;
  // printf("blockIdx.y %d, blockIdx.x %d, thredIdx.x %d\n", blockIdx.y, blockIdx.x, threadIdx.x);

  const int row_size =
        fminf(n_boxes1 - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        fminf(n_boxes2 - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3];
    // printf("boxes2 [%f, %f, %f, %f]\n", boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0],
      // boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1], boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2],
      // boxes2[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3]);
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int box1_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *box1 = boxes1 + box1_idx * 4;
    int i = 0;
    int start = 0;
    // if (row_start == col_start) {
    //   start = threadIdx.x + 1;
    // }
    for (i = start; i < col_size; i++) {
      const int box2_idx = threadsPerBlock * col_start + i;
      const int index = box1_idx * n_boxes2 + box2_idx;
      // printf("box1 index %d, box2 index %d\n", box1_idx, box2_idx);
      overlap[index] = devIoU(box1, block_boxes + i * 4);
    }
  }
}


void _overlap(int boxes1_num, float* boxes1_flat, int boxes2_num, float* boxes2_flat, float* overlap_flat) {

  dim3 blocks(DIVUP(boxes2_num, threadsPerBlock),
              DIVUP(boxes1_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes1_num, boxes1_flat, boxes2_num, boxes2_flat, overlap_flat);
}

#ifdef __cplusplus
}
#endif
